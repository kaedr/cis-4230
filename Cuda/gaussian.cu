#include "hip/hip_runtime.h"
/*!
 * \file   gaussian.c
 * \brief  A Gaussian Elimination solver.
 * \author (C) Copyright 2024 by Peter Chapin <pchapin@vermontstate.edu>
 *
 * This is the serial version of the algorithm.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "gaussian.h"

#define PRIVATE static
#define PUBLIC

__global__ void elimination_kernel( size_t size, floating_type *a, floating_type *b, size_t i ) {
    size_t         k;
    floating_type  m;

    int my_id = threadIdx.x;
    int j = i + 1 + my_id;
    // if (j >= size) {
    //     printf("Ignoring j=%d\n", j);
    //     return;
    // }
    // Synchronize
    // __syncthreads();
    // Subtract multiples of row i from subsequent rows.
    // for( j = i + 1; j < size; ++j ) {
    //     if (!j % my_id == 0) {
    //         continue;
    //     }
    //     m = MATRIX_GET( a, size, j, i ) / MATRIX_GET( a, size, i, i );
    //     for( k = 0; k < size; ++k )
    //         MATRIX_PUT( a, size, j, k, MATRIX_GET( a, size, j, k ) - m * MATRIX_GET( a, size, i, k ) );
    //     b[j] -= m * b[i];
    // }
    // printf("i: %ld, j: %d my_id: %d\n", i, j, my_id);
    m = MATRIX_GET( a, size, j, i ) / MATRIX_GET( a, size, i, i );
    for( k = 0; k < size; ++k )
        MATRIX_PUT( a, size, j, k, MATRIX_GET( a, size, j, k ) - m * MATRIX_GET( a, size, i, k ) );
    b[j] -= m * b[i];

}

//! Does the elimination step of reducing the system. O(n^3)
PRIVATE enum GaussianResult elimination( size_t size, floating_type *a, floating_type *b )
{
    floating_type *temp_array = (floating_type *)malloc( size * sizeof(floating_type) );
    size_t         i, j, k;
    floating_type  temp, m;

    floating_type *dev_a = MATRIX_MAKE( size );
    floating_type *dev_b = (floating_type *)malloc( size * sizeof( floating_type ) );
    hipMalloc( (void **)&dev_a, size * size * sizeof(double) );
    hipMalloc( (void **)&dev_b, size * sizeof(double) );

    for( i = 0; i < size - 1; ++i ) {

        // Find the row with the largest value of |a[j][i]|, j = i, ..., n - 1
        k = i;
        m = fabs( MATRIX_GET( a, size, i, i ) );
        for( j = i + 1; j < size; ++j ) {
            if( fabs( MATRIX_GET( a, size, j, i ) ) > m ) {
                k = j;
                m = fabs( MATRIX_GET( a, size, j, i ) );
            }
        }

        // Check for |a[k][i]| zero.
        // TODO: The value 1.0E-6 is arbitrary. A more disciplined value should be used.
        if( fabs( MATRIX_GET( a, size, k, i ) ) <= 1.0E-6 ) {
            free( temp_array );
            return gaussian_degenerate;
        }

        // Exchange row i and row k, if necessary.
        if( k != i ) {
            memcpy( temp_array, MATRIX_GET_ROW( a, size, i ), size * sizeof( floating_type ) );
            memcpy( MATRIX_GET_ROW( a, size, i ), MATRIX_GET_ROW( a, size, k ), size * sizeof( floating_type ) );
            memcpy( MATRIX_GET_ROW( a, size, k ), temp_array, size * sizeof( floating_type ) );

            // Exchange corresponding elements of b.
            temp = b[i];
            b[i] = b[k];
            b[k] = temp;
        }

        // Subtract multiples of row i from subsequent rows.
        // for( j = i + 1; j < size; ++j ) {
        //     m = MATRIX_GET( a, size, j, i ) / MATRIX_GET( a, size, i, i );
        //     for( k = 0; k < size; ++k )
        //         MATRIX_PUT( a, size, j, k, MATRIX_GET( a, size, j, k ) - m * MATRIX_GET( a, size, i, k ) );
        //     b[j] -= m * b[i];
        // }

        hipMemcpy( dev_a, a, size * size * sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy( dev_b, b, size * sizeof(double), hipMemcpyHostToDevice );
        elimination_kernel<<<1, size - 1 - i>>>( size, dev_a, dev_b, i );
        hipMemcpy( a, dev_a, size * size * sizeof(double), hipMemcpyDeviceToHost );
        hipMemcpy( b, dev_b, size * sizeof(double), hipMemcpyDeviceToHost );
    }
    free( temp_array );
    hipFree( dev_a );
    hipFree( dev_b );
    return gaussian_success;
}

//! Does the back substitution step of solving the system. O(n^2)
PRIVATE enum GaussianResult back_substitution( size_t size, floating_type *a, floating_type *b )
{
    floating_type sum;
    size_t        i, j;
    size_t        counter;

    // We can't count i down from size - 1 to zero (inclusive) because it is unsigned.
    for( counter = 0; counter < size; ++counter ) {
        i = ( size - 1 ) - counter;
        // TODO: The value 1.0E-6 is arbitrary. A more disciplined value should be used.
        if( fabs( MATRIX_GET( a, size, i, i ) ) <= 1.0E-6 ) {
            return gaussian_degenerate;
        }

        sum = b[i];
        for( j = i + 1; j < size; ++j ) {
            sum -= MATRIX_GET( a, size, i, j ) * b[j];
        }
        b[i] = sum / MATRIX_GET( a, size, i, i );
    }
    return gaussian_success;
}


PUBLIC enum GaussianResult gaussian_solve( size_t size, floating_type *a, floating_type *b )
{
    // We can deal with a 1x1 system, but not an empty system.
    if( size == 0 ) return gaussian_error;

    enum GaussianResult return_code = elimination( size, a, b );
    if( return_code == gaussian_success )
        return_code = back_substitution( size, a, b );
    return return_code;
}
