#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "Timer.h"

// Change this type alias to change the data type of the matrix elements.
typedef double floating_type;

enum GaussianResult {
    gaussian_success,     // The system was solved normally.
    gaussian_error,       // A problem with the parameters was detected.
    gaussian_degenerate   // The system is degenerate and does not have a unique solution.
};

//! Does the back substitution step of solving the system. O(n^2)
enum GaussianResult back_substitution( size_t size, floating_type (* __restrict__ a)[size], floating_type * __restrict__ b )
{
    floating_type sum;
    size_t        i, j;
    size_t        counter;

    // We can't count i down from size - 1 to zero (inclusive) because it is unsigned.
    for( counter = 0; counter < size; ++counter ) {
        i = ( size - 1 ) - counter;
        // TODO: The value 1.0E-6 is arbitrary. A more disciplined value should be used.
        if( fabs( a[i][i] ) <= 1.0E-6 ) {
            return gaussian_degenerate;
        }

        sum = b[i];
        for( j = i + 1; j < size; ++j ) {
            sum -= a[i][j] * b[j];
        }
        b[i] = sum / a[i][i];
    }
    return gaussian_success;
}

enum GaussianResult gaussian_solve( size_t size, floating_type (* __restrict__ a)[size], floating_type * __restrict__ b )
{
    // We can deal with a 1x1 system, but not an empty system.
    if( size == 0 ) return gaussian_error;
    enum GaussianResult return_code = elimination( size, a, b );

    if( return_code == gaussian_success )
        return_code = back_substitution( size, a, b );
    return return_code;
}

int main( int argc, char *argv[] )
{
    FILE   *input_file;
    size_t  size;

    if( argc < 2 ) {
        printf( "Error: Expected the name of a system definition file.\n" );
        return EXIT_FAILURE;
    }

    // Open the file.
    if( (input_file = fopen( argv[1], "r" )) == NULL ) {
        printf("Error: Can not open the system definition file.\n");
        return EXIT_FAILURE;
    }

    // Get the size.
    fscanf( input_file, "%zu", &size );

    // Allocate the arrays on the stack... except this overflows the stack for large systems.
    //floating_type a[size][size];
    //floating_type b[size];

    // Allocate the arrays dynamically.
    typedef floating_type row_t[size];
    row_t *a = (row_t *)malloc( size * size * sizeof( floating_type ) );
    floating_type *b = (floating_type *)malloc( size * sizeof( floating_type ) );

    // Get coefficients.
    // Note that the format specifier used here, `%lf`, assumes the matrix elements have type
    // double. See the declaration of `floating_type` at the top of gaussian.h.
    //
    for( size_t i = 0; i < size; ++i ) {
        for( size_t j = 0; j < size; ++j ) {
            fscanf( input_file, "%lf", &a[i][j] );
        }
        fscanf( input_file, "%lf", &b[i] );
    }
    fclose( input_file );

    // Do the calculations.
    Timer stopwatch;
    Timer_initialize( &stopwatch );
    Timer_start( &stopwatch );
    enum GaussianResult result = gaussian_solve( size, a, b );
    Timer_stop( &stopwatch );

    // Display the results.
    switch( result ) {
    case gaussian_success:
        // printf( "\nSolution is\n" );
        // for( size_t i = 0; i < size; ++i ) {
        //     printf( " x[%4zu] = %9.5f\n", i, b[i] );
        // }
        printf( "Execution time = %ld milliseconds\n", Timer_time( &stopwatch ) );
        break;

    case gaussian_error:
        printf( "Parameter problem in call to gaussian_solve( )\n" );
        break;

    case gaussian_degenerate:
        printf( "System is degenerate. It does not have a unique solution.\n" );
        break;
    }

    // Clean up the dynamically allocated space.
    free( a );
    free( b );
    return EXIT_SUCCESS;
}
